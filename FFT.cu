#include "hip/hip_runtime.h"
//The Code for Fast Fourier Transforms

#define N 64

#include "timerc.h"
#include <stdio.h>
#include <complex.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_complex.h>

__host__ __device__ hipDoubleComplex mult(hipDoubleComplex a, hipDoubleComplex b)
{
	double a1 = hipCreal(a) * hipCreal(b) - hipCimag(a) * hipCimag(b);
	double a2 = hipCreal(a) * hipCimag(b) + hipCreal(b) * hipCimag(a);
	return make_hipDoubleComplex(a1, a2);
}

__host__ __device__ hipDoubleComplex plus(hipDoubleComplex a, hipDoubleComplex b)
{
	double a1 = hipCreal(a) + hipCreal(b);
	double a2 = hipCimag(a) + hipCimag(b);
	return make_hipDoubleComplex(a1, a2);

}

__host__ __device__ hipDoubleComplex minus(hipDoubleComplex a, hipDoubleComplex b)
{
	double a1 = hipCreal(a) - hipCreal(b);
	double a2 = hipCimag(a) - hipCimag(b);
	return make_hipDoubleComplex(a1, a2);

}

__host__ __device__ hipDoubleComplex pwcom(hipDoubleComplex a, int b)
{
	hipDoubleComplex ori = make_hipDoubleComplex(1.0, 0.0);
	for(int i = 0; i < b; i++)
		ori = mult(ori, a);
	return ori;
	
}

__host__ __device__ int comp(hipDoubleComplex *vec1, hipDoubleComplex *vec2, int n)
{
	int error = 0;
	for(int i = 0; i < n; i++)
		if(((int)hipCreal(vec1[i]) != (int)hipCreal(vec2[i])) || ((int)hipCimag(vec1[i]) != (int)hipCimag(vec2[i])))
			error += 1;
	return error;

}

__global__ void cumerge(hipDoubleComplex* in, int n)
{
	int half = n/2;
	hipDoubleComplex temp;
	temp = in[threadIdx.x];
	__syncthreads();
	if(threadIdx.x < half)
		in[threadIdx.x * 2] = temp;
	else
		in[(threadIdx.x - half) * 2 + 1] = temp;
}

//Dyanmic Parallelism
__global__ void dynamic(hipDoubleComplex* in, hipDoubleComplex w, int n, int deg, int start)
{
	hipDoubleComplex temp1, temp2;	
	temp1 = in[threadIdx.x + start];
	temp2 = in[threadIdx.x + start + n/2];
	in[threadIdx.x + start] = plus(temp1, temp2);
	in[threadIdx.x + start + n/2] = mult(pwcom(w, threadIdx.x * deg), minus(temp1, temp2));
	__syncthreads();
	if(threadIdx.x == 0 && n >= 4){
		dynamic<<<1, n/4>>>(in, w, n/2, deg*2, start);
		dynamic<<<1, n/4>>>(in, w, n/2, deg*2, start + n/2);
		hipDeviceSynchronize();
		cumerge<<<1, n>>>(&in[start], n);
	}
}

__global__ void rearrange(hipDoubleComplex* in, int n)
{
	for(int i = 4; i <= n; i *= 2){
		hipDoubleComplex temp;
		temp = in[threadIdx.x];
		__syncthreads();
		if(threadIdx.x % i < i/2)
			in[threadIdx.x + threadIdx.x % i] = temp;
		else
			in[threadIdx.x - i/2 + threadIdx.x % (i/2) + 1] = temp;
		__syncthreads();
	}
}

__global__ void one_kernel(hipDoubleComplex* in, hipDoubleComplex w, int n)
{
	__shared__ hipDoubleComplex temp[N];
	temp[threadIdx.x] = in[threadIdx.x];
	temp[threadIdx.x + n/2] = in[threadIdx.x + n/2];
	__syncthreads();	
	int degree = 1;
	for(int i = n; i >= 2; i/=2){
		int half = i/2;
		int id = threadIdx.x + (threadIdx.x / half) * half;
		hipDoubleComplex temp1, temp2;
		temp1 = temp[id];
		temp2 = temp[id + half];
		temp[id] = plus(temp1, temp2);
		temp[id + half] = mult(pwcom(w, ((threadIdx.x % half) * degree)), minus(temp1, temp2));
		degree *= 2;
		__syncthreads();
	}
	in[threadIdx.x] = temp[threadIdx.x];
	in[threadIdx.x + n/2] = temp[threadIdx.x + n/2];
	__syncthreads();	
}

void comb(hipDoubleComplex *res, int n, int a, int b)
{
	hipDoubleComplex re1[n], re2[n];
	for(int i = 0; i < n; i++){
		re1[i] = res[a + i];
		re2[i] = res[b + i];
	}	
	for(int i = 0; i < 2 * n; i += 2){
		res[i] = re1[i/2];
		res[i + 1] = re2[i/2];
	}
}


void trans(hipDoubleComplex *a, int n, int deg, hipDoubleComplex w)
{	
	hipDoubleComplex t;	
	if(n == 2){
			t = a[0];
			a[0] = plus(a[0], a[1]); 
			a[1] = minus(t, a[1]);			
	}
	else{
		hipDoubleComplex temp1, temp2;
		for(int i = 0; i < n / 2; i++){
			temp1 = a[i];
			temp2 = a[i + n/2];
			a[i] = plus(temp1, temp2);
			a[n/2 + i] = mult(pwcom(w, i * deg), minus(temp1, temp2));
		}
/*
		printf("Test\n");
		for(int i = 0; i < n; i++)
			printf("%.0f + %.0fi\n", hipCreal(a[i]), hipCimag(a[i]));
		printf("Test finished\n");
*/
		trans(&a[0], n/2, deg*2, w);
		trans(&a[n/2], n/2, deg*2, w);
		comb(a, n/2, 0, n/2);
	}
}

hipDoubleComplex calc (int n)
{
	double theta = 2 * M_PI / (double)n;
	hipDoubleComplex w = make_hipDoubleComplex(cos(theta), sin(theta));	
	return w;
}

int main()
{
	float ctime, gtime;
	srand(time(NULL));
	hipDoubleComplex *vec_host = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * N);
	hipDoubleComplex *vec_host_out = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * N);
	hipDoubleComplex *vec_dev;
	hipMalloc(&vec_dev, sizeof(hipDoubleComplex) * N);
	for(int i = 0; i < N; i++)
		vec_host[i] = make_hipDoubleComplex((double)(rand()%10), (double)(rand()%10));
/*
	printf("Vector:\n");
	for(int i = 0; i < N; i++)
		printf("%.0f + %.0fi\n", hipCreal(vec_host[i]), hipCimag(vec_host[i]));
	printf("\n\n");
*/
	hipMemcpy(vec_dev, vec_host, sizeof(hipDoubleComplex) * N, hipMemcpyHostToDevice);
	gstart();
	//dynamic<<<1, N/2>>>(vec_dev, make_hipDoubleComplex(0.0, 1.0), N, 1, 0);
	//dynamic<<<1, N/2>>>(vec_dev, calc(N), N, 1, 0);
	one_kernel<<<1, N/2>>>(vec_dev, make_hipDoubleComplex(0.0, 1.0), N);
	//one_kernel<<<1, N/2>>>(vec_dev, calc(N), N);
	rearrange<<<1, N>>>(vec_dev, N);
	gend(&gtime);
	hipMemcpy(vec_host_out, vec_dev, sizeof(hipDoubleComplex) * N, hipMemcpyDeviceToHost);
	cstart();
	trans(vec_host, N, 1, make_hipDoubleComplex(0.0, 1.0));
	//trans(vec_host, N, 1, calc(N));
	cend(&ctime);
/*
	printf("GPU:\n");
	for(int i = 0; i < N; i++)
		printf("%.0f + %.0fi\n", hipCreal(vec_host_out[i]), hipCimag(vec_host_out[i]));
	printf("CPU:\n");
	for(int i = 0; i < N; i++)
		printf("%.0f + %.0fi\n", hipCreal(vec_host[i]), hipCimag(vec_host[i]));
*/
	if(comp(vec_host, vec_host_out, N) == 0)
		printf("Two vectors computed by CPU and GPU matches!\n");
	else 
		printf("Two vectors computed by CPU and GPU does not match!\n");
	printf("CPU Time: %f GPU Time: %f\n", ctime, gtime);
	free(vec_host);
	free(vec_host_out);
	hipFree(vec_dev);
	return 0;
}
