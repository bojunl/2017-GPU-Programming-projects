#include "hip/hip_runtime.h"
//Code for Discrete Fourier Transform

#define N 64

#include "timerc.h"
#include <stdio.h>
#include <complex.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_complex.h>

//the multiplication operator in hip/hip_complex.h library is hipCmul, but it seems to be problematic because it always gives meaningless result, so I decided to rewrite the operator
__host__ __device__ hipDoubleComplex mult(hipDoubleComplex a, hipDoubleComplex b)
{
	double a1 = hipCreal(a) * hipCreal(b) - hipCimag(a) * hipCimag(b);
	double a2 = hipCreal(a) * hipCimag(b) + hipCreal(b) * hipCimag(a);
	return make_hipDoubleComplex(a1, a2);
}

__host__ __device__ hipDoubleComplex plus(hipDoubleComplex a, hipDoubleComplex b)
{
	double a1 = hipCreal(a) + hipCreal(b);
	double a2 = hipCimag(a) + hipCimag(b);
	return make_hipDoubleComplex(a1, a2);

}

__host__ __device__ int comp(hipDoubleComplex *vec1, hipDoubleComplex *vec2, int n)
{
	int error = 0;
	for(int i = 0; i < n; i++)
		if(((int)hipCreal(vec1[i]) != (int)hipCreal(vec2[i])) || ((int)hipCimag(vec1[i]) != (int)hipCimag(vec2[i])))
			error += 1;
	return error;

}

__global__ void naive_mult(hipDoubleComplex *matrix, hipDoubleComplex *vector, int n)
{
	matrix[threadIdx.x * n + blockIdx.x] = mult(matrix[threadIdx.x * n + blockIdx.x], vector[blockIdx.x]);
}



__global__ void naive_sum(hipDoubleComplex *matrix_in, hipDoubleComplex *vector_out, int n)
{
	int id = n * blockIdx.x + threadIdx.x;
	int space = n/2;
	while(space >= 1){
		if((id % n + space) < n){
			matrix_in[id + space] = hipCadd(matrix_in[id + space], matrix_in[id]);	
			__syncthreads();		
		}
		id += space;
		space /= 2;	
	}
	if(threadIdx.x == 0) 
		vector_out[blockIdx.x] = matrix_in[blockIdx.x * n + n - 1];
}

__global__ void naive_sum_shared(hipDoubleComplex *matrix_in, hipDoubleComplex *vector_out, int n)
{
	__shared__ hipDoubleComplex temp[N];
	int id = n * blockIdx.x + threadIdx.x;
	int space = n/2;
	if(threadIdx.x * 2 < n){
		temp[threadIdx.x] = matrix_in[id];
		temp[threadIdx.x + space] = matrix_in[id + space];	
	}
	__syncthreads();
	while(space >= 1){
		if((id % n + space) < n){
			temp[id % n + space] = hipCadd(temp[id % n + space], temp[id % n]);
			__syncthreads();
		}
		id += space;
		space /= 2;
	}
	if(threadIdx.x == 0)
		vector_out[blockIdx.x] = temp[n-1];
}

hipDoubleComplex calc (int p, int n)
{
	double theta = 2 * M_PI / (double)n;
	hipDoubleComplex ori = make_hipDoubleComplex(1.0, 0.0); 
	hipDoubleComplex w = make_hipDoubleComplex(cos(theta), sin(theta));	
 	for(int i = 1; i <= p; i++)
		ori = mult(ori, w);
	return ori;
}

int main()
{
	float host_time, dev_time;	
	srand(time(NULL));
	hipDoubleComplex *dev_m, *m = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * N * N);
	
	hipMalloc(&dev_m, sizeof(hipDoubleComplex) * N * N);
	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++)
			m[N * i + j] = calc(i*j, N);
	hipMemcpy(dev_m, m, sizeof(hipDoubleComplex) * N * N, hipMemcpyHostToDevice);

/*
	hipDoubleComplex *m2 = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * N * N);
	hipMemcpy(m2, dev_m, sizeof(hipDoubleComplex) * N * N, hipMemcpyDeviceToHost);
	printf("\ncopied Transformation matrix is \n");
	for(int i = 0; i < N; i++){
		printf("\n");		
		for(int j = 0; j < N; j++)
				printf("%.0f + %.0fi ", hipCreal(m2[N * i + j]), hipCimag(m2[N * i + j]));
	}
	printf("\n\n\n");
*/

	hipDoubleComplex *vec_in = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * N);
	hipDoubleComplex *vec_out = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * N);
	hipDoubleComplex *dev_vec_in;
	hipDoubleComplex *dev_vec_out;
	hipMalloc(&dev_vec_in, sizeof(hipDoubleComplex) * N);
	hipMalloc(&dev_vec_out, sizeof(hipDoubleComplex) * N);
	hipDoubleComplex *vec_out2 = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * N);
	for(int i = 0; i < N; i++){
		vec_out[i] = make_hipDoubleComplex(0.0, 0.0);
		vec_in[i] = make_hipDoubleComplex((double)(rand()%10), (double)(rand()%10));}

/*	
	printf("\nrandomly generated vector is\n");
	for(int i = 0; i < N; i++)
		printf("%.0f + %.0fi\n", hipCreal(vec_in[i]), hipCimag(vec_in[i]));
	printf("\n\n");
*/


	hipMemcpy(dev_vec_in, vec_in, sizeof(hipDoubleComplex) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_vec_out, vec_out, sizeof(hipDoubleComplex) * N, hipMemcpyHostToDevice);


	cstart();
	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++)
			vec_out[i] =hipCadd(vec_out[i], hipCmul(m[i * N + j], vec_in[j]));
	cend(&host_time);

	gstart();
	naive_mult<<<N, N>>>(dev_m, dev_vec_in, N);
/*	
	printf("\nafter multiplication \n");
	hipDoubleComplex *dev_m_2 = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * N * N);	
	hipMemcpy(dev_m_2, dev_m, sizeof(hipDoubleComplex) * N * N, hipMemcpyDeviceToHost);
	for(int i = 0; i < N; i++){
		printf("\n");		
		for(int j = 0; j < N; j++)
				printf("%.0f + %.0fi \n\n", hipCreal(dev_m_2[N * i + j]), hipCimag(dev_m_2[N * i + j]));
	}
	printf("\n\n");
*/
	
	naive_sum<<<N, N/2>>>(dev_m, dev_vec_out, N);
//	naive_sum_shared<<<N, N/2>>>(dev_m, dev_vec_out, N);
	gend(&dev_time);
	hipMemcpy(vec_out2, dev_vec_out, sizeof(hipDoubleComplex) * N, hipMemcpyDeviceToHost);
/*	
	// Testing the output of the original complex-valued matrix

	for(int i = 0; i < N; i++){
		printf("\n");		
		for(int j = 0; j < N; j++)
			if(cimag(m[N * i + j]))
				printf("%.0fi ", cimag(m[N * i + j]));
			else 
				printf("%.0f ", creal(m[N * i + j]));
	}
*/	
/*
	for(int i = 0; i < N; i++)
		printf("%.0f + %.0fi\n", hipCreal(vec_out2[i]), hipCimag(vec_out2[i]));
	printf("\n\n\n");
	for(int i = 0; i < N; i++)
		printf("%.0f + %.0fi\n", hipCreal(vec_out[i]), hipCimag(vec_out[i]));
*/	
	if(comp(vec_out, vec_out2, N) == 0)
		printf("Two vectors computed by CPU and GPU matches!\n");
	else 
		printf("Two vectors computed by CPU and GPU does not match!\n");
	printf("CPU Time: %f\n GPU Time: %f\n", host_time, dev_time);

	free(m);
	free(vec_in);
	free(vec_out);
	free(vec_out2);
	hipFree(dev_m);
	hipFree(dev_vec_in);
	hipFree(dev_vec_out);	
	return 0;
}
